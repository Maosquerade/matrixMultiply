#include <stdio.h>
#include <hip/hip_runtime.h>

void printDeviceProp(hipDeviceProp_t prop) {
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %d.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %d.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);
    printf("textureAlignment : %d.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);


}
int main() {
    int count;

    // 取得支持 cuda 的设备数目
    hipGetDeviceCount(&count);

    if (count == 0) {
        printf("There is no device.\n");
        return -1;
    }

    for (int i = 0; i< count; i += 1) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        //打印设备信息
        printDeviceProp(prop);

        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                break;
            }
        }
    }
} 
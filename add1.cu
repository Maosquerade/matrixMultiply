#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void add( int a, int b, int *c ) {
    *c = a + b;
}
int main( void ) {
    int c;
    int *dev_c;
    //cudaMalloc()
    hipMalloc( (void**)&dev_c, sizeof(int) );
    //核函数执行
    add<<<1,1>>>( 2, 7, dev_c );   
    //cudaMemcpy()
    hipMemcpy( &c, dev_c, sizeof(int),hipMemcpyDeviceToHost ) ;
    printf( "2 + 7 = %d\n", c );
    //cudaFree()
    hipFree( dev_c );
 
    return 0;
}
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int *c) {
    *c *= 2;
    printf("Hello, World!\n");
}

int main(void) {
    int c = 2;
    int *dev_c;
    //cudaMalloc()
    hipMalloc(&dev_c, sizeof(int));
    hipMemcpy(dev_c, &c, sizeof(int), hipMemcpyHostToDevice);
    //核函数执行
    add<<<1,1>>>(dev_c);
    //cudaMemcpy()
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("2 * 2 = %d\n", c);
    hipFree(dev_c);

    return 0;
}
